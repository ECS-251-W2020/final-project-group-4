#include <hip/hip_runtime.h>

__global__ void double_kernel(const float* data, float* output, size_t n) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        output[idx] = data[idx] * 2;
    }
}

void launch_double_ext_cuda_kernel(const float* data, float* output, size_t n){
    const int threads = 256;
    const int blocks = (n + threads - 1) / threads;
    double_kernel<<<blocks, threads>>>(data, output, n);
}
