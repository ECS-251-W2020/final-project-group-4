
#include "hip/hip_runtime.h"


__global__ void double_kernel(const float* data, float* output, size_t n) {
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < n) {
		output[idx] = data[idx] * 2;
	}
}

__declspec(dllexport) void launch_double_ext_cuda_kernel(const float* data, float* output, size_t n) {
	const int threads = 256;
	const int blocks = (n + threads - 1) / threads;
	double_kernel <<<blocks, threads >>> (data, output, n);
}


