#include "hip/hip_runtime.h"
#include "CUDA_runtime.h"
#include <>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "constant.h"
typedef short WORD;
typedef int DWORD;
typedef int LONG;

// sbox used in host
const unsigned char box[256] = {
	// 0     1     2     3     4     5     6     7     8     9     a     b     c     d     e     f
	0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76, //  0
	0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0, //  1
	0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15, //  2
	0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75, //  3
	0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84, //  4
	0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf, //  5
	0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8, //  6
	0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2, //  7
	0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73, //  8
	0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb, //  9
	0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79, //  a
	0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08, //  b
	0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a, //  c
	0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e, //  d
	0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf, //  e
	0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16 };//  f

// Round Keys
const unsigned char rcon[10] = {
	0x01, 0x02, 0x04, 0x08, 0x10,
	0x20, 0x40, 0x80, 0x1b, 0x36 };


// sbox used in device
__device__ static unsigned char s_box[256] = {
	// 0     1     2     3     4     5     6     7     8     9     a     b     c     d     e     f
	0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76, //  0
	0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0, //  1
	0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15, //  2
	0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75, //  3
	0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84, //  4
	0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf, //  5
	0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8, //  6
	0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2, //  7
	0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73, //  8
	0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb, //  9
	0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79, //  a
	0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08, //  b
	0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a, //  c
	0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e, //  d
	0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf, //  e
	0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16 };//  f

// inversed sbox used in device
__device__ static unsigned char inv_s_box[256] = {
	// 0     1     2     3     4     5     6     7     8     9     a     b     c     d     e     f
	0x52, 0x09, 0x6a, 0xd5, 0x30, 0x36, 0xa5, 0x38, 0xbf, 0x40, 0xa3, 0x9e, 0x81, 0xf3, 0xd7, 0xfb, //  0
	0x7c, 0xe3, 0x39, 0x82, 0x9b, 0x2f, 0xff, 0x87, 0x34, 0x8e, 0x43, 0x44, 0xc4, 0xde, 0xe9, 0xcb, //  1
	0x54, 0x7b, 0x94, 0x32, 0xa6, 0xc2, 0x23, 0x3d, 0xee, 0x4c, 0x95, 0x0b, 0x42, 0xfa, 0xc3, 0x4e, //  2
	0x08, 0x2e, 0xa1, 0x66, 0x28, 0xd9, 0x24, 0xb2, 0x76, 0x5b, 0xa2, 0x49, 0x6d, 0x8b, 0xd1, 0x25, //  3
	0x72, 0xf8, 0xf6, 0x64, 0x86, 0x68, 0x98, 0x16, 0xd4, 0xa4, 0x5c, 0xcc, 0x5d, 0x65, 0xb6, 0x92, //  4
	0x6c, 0x70, 0x48, 0x50, 0xfd, 0xed, 0xb9, 0xda, 0x5e, 0x15, 0x46, 0x57, 0xa7, 0x8d, 0x9d, 0x84, //  5
	0x90, 0xd8, 0xab, 0x00, 0x8c, 0xbc, 0xd3, 0x0a, 0xf7, 0xe4, 0x58, 0x05, 0xb8, 0xb3, 0x45, 0x06, //  6
	0xd0, 0x2c, 0x1e, 0x8f, 0xca, 0x3f, 0x0f, 0x02, 0xc1, 0xaf, 0xbd, 0x03, 0x01, 0x13, 0x8a, 0x6b, //  7
	0x3a, 0x91, 0x11, 0x41, 0x4f, 0x67, 0xdc, 0xea, 0x97, 0xf2, 0xcf, 0xce, 0xf0, 0xb4, 0xe6, 0x73, //  8
	0x96, 0xac, 0x74, 0x22, 0xe7, 0xad, 0x35, 0x85, 0xe2, 0xf9, 0x37, 0xe8, 0x1c, 0x75, 0xdf, 0x6e, //  9
	0x47, 0xf1, 0x1a, 0x71, 0x1d, 0x29, 0xc5, 0x89, 0x6f, 0xb7, 0x62, 0x0e, 0xaa, 0x18, 0xbe, 0x1b, //  a
	0xfc, 0x56, 0x3e, 0x4b, 0xc6, 0xd2, 0x79, 0x20, 0x9a, 0xdb, 0xc0, 0xfe, 0x78, 0xcd, 0x5a, 0xf4, //  b
	0x1f, 0xdd, 0xa8, 0x33, 0x88, 0x07, 0xc7, 0x31, 0xb1, 0x12, 0x10, 0x59, 0x27, 0x80, 0xec, 0x5f, //  c
	0x60, 0x51, 0x7f, 0xa9, 0x19, 0xb5, 0x4a, 0x0d, 0x2d, 0xe5, 0x7a, 0x9f, 0x93, 0xc9, 0x9c, 0xef, //  d
	0xa0, 0xe0, 0x3b, 0x4d, 0xae, 0x2a, 0xf5, 0xb0, 0xc8, 0xeb, 0xbb, 0x3c, 0x83, 0x53, 0x99, 0x61, //  e
	0x17, 0x2b, 0x04, 0x7e, 0xba, 0x77, 0xd6, 0x26, 0xe1, 0x69, 0x14, 0x63, 0x55, 0x21, 0x0c, 0x7d };//  f

__device__ const int Nr = AES_NR; // numbers of rounds
__device__ const int Nk = AES_NK;  // numbers of columns in a key
__device__ const int Nb = AES_NB; // key size


__device__ void shift_rows(unsigned char* state) {
	unsigned char i, k, s, col;
	for (i = 1; i < 4; i++) {
		s = 0;
		while (s < i) {
			col = state[Nk * i + 0];

			for (k = 1; k < Nk; k++) {
				state[Nk * i + k - 1] = state[Nk * i + k];
			}

			state[Nk * i + Nk - 1] = col;
			s++;
		}
	}
}


__device__ void inv_shift_rows(unsigned char* state) {
	unsigned char i, k, s, col;
	for (i = 1; i < 4; i++) {
		s = 0;
		while (s < i) {
			col = state[Nk * i + Nk - 1];

			for (k = Nk - 1; k > 0; k--) {
				state[Nk * i + k] = state[Nk * i + k - 1];
			}

			state[Nk * i + 0] = col;
			s++;
		}
	}
}


__device__ unsigned char gmult(unsigned char a, unsigned char b) {

	unsigned char p = 0, i = 0, hbs = 0;

	for (i = 0; i < 8; i++) {
		if (b & 1) {
			p ^= a;
		}

		hbs = a & 0x80;
		a <<= 1;
		if (hbs) a ^= 0x1b;
		b >>= 1;
	}

	return (unsigned char)p;
}


__device__ void coef_mult(unsigned char* a, unsigned char* b, unsigned char* d) {

	d[0] = gmult(a[0], b[0]) ^ gmult(a[3], b[1]) ^ gmult(a[2], b[2]) ^ gmult(a[1], b[3]);
	d[1] = gmult(a[1], b[0]) ^ gmult(a[0], b[1]) ^ gmult(a[3], b[2]) ^ gmult(a[2], b[3]);
	d[2] = gmult(a[2], b[0]) ^ gmult(a[1], b[1]) ^ gmult(a[0], b[2]) ^ gmult(a[3], b[3]);
	d[3] = gmult(a[3], b[0]) ^ gmult(a[2], b[1]) ^ gmult(a[1], b[2]) ^ gmult(a[0], b[3]);
}


__device__ void mix_columns(unsigned char* state) {

	unsigned char a[] = { 0x02, 0x01, 0x01, 0x03 };
	unsigned char i, j, col[4], res[4];

	for (j = 0; j < Nk; j++) {
		for (i = 0; i < 4; i++) {
			col[i] = state[Nk * i + j];
		}

		coef_mult(a, col, res);

		for (i = 0; i < 4; i++) {
			state[Nk * i + j] = res[i];
		}
	}
}

__device__ void inv_mix_columns(unsigned char* state) {

	unsigned char a[] = { 0x0e, 0x09, 0x0d, 0x0b };
	unsigned char i, j, col[4], res[4];

	for (j = 0; j < Nk; j++) {
		for (i = 0; i < 4; i++) {
			col[i] = state[Nk * i + j];
		}

		coef_mult(a, col, res);

		for (i = 0; i < 4; i++) {
			state[Nk * i + j] = res[i];
		}
	}
}

// expand original key so to use it in AddRoundKey stage - key_xor()
void key_expansion(unsigned char* key, unsigned char* w) {

	unsigned char r, i, j, k, col[4];
	col[0] = 0; col[1] = 0; col[2] = 0; col[3] = 0;
	// first round key is just the key
	for (j = 0; j < Nk; j++) {
		for (i = 0; i < 4; i++) {
			w[Nk * i + j] = key[Nk * i + j];
		}
	}

	for (r = 1; r < Nr + 1; r++) {
		for (j = 0; j < Nk; j++) {
			for (i = 0; i < 4; i++) {
				if (j % Nk != 0) {
					col[i] = w[r * Nb + Nk * i + j - 1];
				}
				else {
					col[i] = w[(r - 1) * Nb + Nk * i + Nk - 1];
				}
			}

			if (j % Nk == 0) {
				// rotate 4 bytes in word
				k = col[0];
				col[0] = col[1];
				col[1] = col[2];
				col[2] = col[3];
				col[3] = k;

				col[0] = box[col[0]];
				col[1] = box[col[1]];
				col[2] = box[col[2]];
				col[3] = box[col[3]];

				col[0] = col[0] ^ rcon[r - 1];
			}

			w[r * Nb + Nk * 0 + j] = w[(r - 1) * Nb + Nk * 0 + j] ^ col[0];
			w[r * Nb + Nk * 1 + j] = w[(r - 1) * Nb + Nk * 1 + j] ^ col[1];
			w[r * Nb + Nk * 2 + j] = w[(r - 1) * Nb + Nk * 2 + j] ^ col[2];
			w[r * Nb + Nk * 3 + j] = w[(r - 1) * Nb + Nk * 3 + j] ^ col[3];
		}
	}
}

__device__ void key_xor(unsigned char* state, unsigned char* key) {
	unsigned char i;
	for (i = 0; i < Nb; i++)
	{
		state[i] = state[i] ^ key[i];
	}
}


#pragma pack(push, 1)
typedef struct tagBITMAPFILEHEADER
{
	WORD bfType;  // specifies the file type
	DWORD bfSize;  // specifies the size in bytes of the bitmap file
	WORD bfReserved1;  // reserved; must be 0
	WORD bfReserved2;  // reserved; must be 0
	DWORD bOffBits;  // species the offset in bytes from the bitmapfileheader to the bitmap bits
}BITMAPFILEHEADER;
#pragma pack(pop)


#pragma pack(push, 1)
typedef struct tagBITMAPINFOHEADER
{
	DWORD biSize;  // specifies the number of bytes required by the struct
	LONG biWidth;  // specifies width in pixels
	LONG biHeight;  // species height in pixels
	WORD biPlanes; // specifies the number of color planes, must be 1
	WORD biBitCount; // specifies the number of bit per pixel
	DWORD biCompression;// spcifies the type of compression
	DWORD biSizeImage;  // size of image in bytes
	LONG biXPelsPerMeter;  // number of pixels per meter in x axis
	LONG biYPelsPerMeter;  // number of pixels per meter in y axis
	DWORD biClrUsed;  // number of colors used by th ebitmap
	DWORD biClrImportant;  // number of colors that are important
}BITMAPINFOHEADER;
#pragma pack(pop)


// load image from file
unsigned char* LoadBitmapFile(char* filename, BITMAPINFOHEADER* bitmapInfoHeader, BITMAPFILEHEADER* bitmapFileHeader)
{
	FILE* filePtr; // our file pointer
	unsigned char* bitmapImage;  // store image data

	// open filename in read binary mode
	filePtr = fopen(filename, "rb");
	if (filePtr == NULL)
		return NULL;

	// read the bitmap file header
	fread(bitmapFileHeader, sizeof(BITMAPFILEHEADER), 1, filePtr);

	// verify that this is a bmp file by check bitmap id
	if (bitmapFileHeader->bfType != 0x4D42)
	{
		fclose(filePtr);
		return NULL;
	}

	// read the bitmap info header
	fread(bitmapInfoHeader, sizeof(BITMAPINFOHEADER), 1, filePtr);

	// move file point to the begging of bitmap data
	fseek(filePtr, long(sizeof(BITMAPFILEHEADER) + sizeof(BITMAPINFOHEADER)), SEEK_SET);

	// allocate enough memory for the bitmap image data
	bitmapImage = (unsigned char*)malloc(bitmapInfoHeader->biSizeImage);

	// verify memory allocation
	if (!bitmapImage)
	{
		free(bitmapImage);
		fclose(filePtr);
		return NULL;
	}

	// read in the bitmap image data
	fread(bitmapImage, 1, bitmapInfoHeader->biSizeImage, filePtr);

	// make sure bitmap image data was read
	if (bitmapImage == NULL)
	{
		fclose(filePtr);
		return NULL;
	}

	unsigned char* d_bitmapImage;  // store image data in device

	// Allocate size to array in device memory
	hipMalloc((void**)&d_bitmapImage, bitmapInfoHeader->biSizeImage);

	// Copy data from host to device
	hipMemcpy(d_bitmapImage, bitmapImage, bitmapInfoHeader->biSizeImage, hipMemcpyHostToDevice);

	// Kernel call
	hipMemcpy(bitmapImage, d_bitmapImage, bitmapInfoHeader->biSizeImage, hipMemcpyDeviceToHost);

	// close file and return bitmap iamge data
	fclose(filePtr);
	return bitmapImage;
}


// Save image to file
void SaveBitmapFile(char* filename, unsigned char* bitmapImage, BITMAPFILEHEADER* bitmapFileHeader, BITMAPINFOHEADER* bitmapInfoHeader)
{
	FILE* filePtr; // our file pointer

	// open filename in write binary mode
	filePtr = fopen(filename, "wb");
	if (filePtr == NULL)
	{
		printf("\nERROR: Cannot open file %s", filename);
		exit(1);
	}

	// write the bitmap file header
	fwrite(bitmapFileHeader, sizeof(BITMAPFILEHEADER), 1, filePtr);

	// write the bitmap info header
	fwrite(bitmapInfoHeader, sizeof(BITMAPINFOHEADER), 1, filePtr);

	// write in the bitmap image data
	fwrite(bitmapImage, bitmapInfoHeader->biSizeImage, 1, filePtr);

	// close file
	fclose(filePtr);
}


__global__ void encrypt(unsigned char* bitmapImage, unsigned char* expanded_key, int size, int threadN)
{
	int threadId = threadIdx.x + blockIdx.x * blockDim.x;
	__shared__ unsigned char sdata[512 * Nb];
	int i;
	unsigned int tid = threadIdx.x;

	for (int k = tid * Nb; k < (tid + 1) * Nb; k++) {
		int gid = k + blockIdx.x * 512 * Nb;
		if (gid < size)
			sdata[k] = bitmapImage[gid];

	}
	__syncthreads();

	// key_xor
	key_xor(&sdata[tid * Nb], &expanded_key[0]);
	__syncthreads();

	for (int r = 1; r < Nr; r++) {
		// substitution
		for (i = tid * Nb; i < (tid + 1) * Nb; i++) {
			sdata[i] = s_box[sdata[i]];
		}
		__syncthreads();

		// shift rows
		shift_rows(&sdata[tid * Nb]);
		__syncthreads();

		// mix columns
		mix_columns(&sdata[tid * Nb]);
		__syncthreads();

		// key_xor
		key_xor(&sdata[tid * Nb], &expanded_key[r * Nb]);
		__syncthreads();
	}

	// substitution
	for (i = tid * Nb; i < (tid + 1) * Nb; i++) {
		sdata[i] = s_box[sdata[i]];
	}
	__syncthreads();

	// shift rows
	shift_rows(&sdata[tid * Nb]);
	__syncthreads();

	// key_xor
	key_xor(&sdata[tid * Nb], &expanded_key[Nr * Nb]);
	__syncthreads();

	for (int k = tid * Nb; k < (tid + 1) * Nb; k++) {
		int gid = k + blockIdx.x * 512 * Nb;
		if (gid < size)
			bitmapImage[gid] = sdata[k];
	}
	__syncthreads();

}


__global__ void decrypt(unsigned char* bitmapImage, unsigned char* expanded_key, int size, int threadN)
{
	int threadId = threadIdx.x + blockIdx.x * blockDim.x;
	__shared__ unsigned char sdata[512 * Nb];
	int i;
	unsigned int tid = threadIdx.x;

	for (int k = tid * Nb; k < (tid + 1) * Nb; k++) {
		int gid = k + blockIdx.x * 512 * Nb;
		if (gid < size)
			sdata[k] = bitmapImage[gid];
	}
	__syncthreads();

	// key_xor
	key_xor(&sdata[tid *  Nb], &expanded_key[Nr *  Nb]);
	__syncthreads();

	for (int r = 1; r < Nr; r++) {
		// shift rows
		inv_shift_rows(&sdata[tid *  Nb]);
		__syncthreads();

		// substitution
		for (i = tid * Nb; i < (tid + 1) * Nb; i++) {
			sdata[i] = inv_s_box[sdata[i]];
		}
		__syncthreads();

		// key_xor
		key_xor(&sdata[tid *  Nb], &expanded_key[(Nr - r) *  Nb]);
		__syncthreads();

		// mix columns
		inv_mix_columns(&sdata[tid *  Nb]);
		__syncthreads();
	}

	// substitution
	for (i = tid * Nb; i < (tid + 1) * Nb; i++) {
		sdata[i] = inv_s_box[sdata[i]];
	}
	__syncthreads();

	// shift rows
	inv_shift_rows(&sdata[tid *  Nb]);
	__syncthreads();

	// key_xor
	key_xor(&sdata[tid * Nb], &expanded_key[0]);
	__syncthreads();

	for (int k = tid * Nb; k < (tid + 1) * Nb; k++) {
		int gid = k + blockIdx.x * 512 * Nb;
		if (gid < size)
			bitmapImage[gid] = sdata[k];
	}
	__syncthreads();

}

__declspec(dllexport) void launchEncryptKernel(unsigned char* d_bitmapImage, unsigned char* d_expanded_key, int size) {
	int B = ceil((double)size / (512 * Nb));
	int T = 512;
	int threadN = B * T;
	encrypt <<<B, T >>> (d_bitmapImage, d_expanded_key, size, threadN);
}
__declspec(dllexport) void launchDecryptKernel(unsigned char* d_bitmapImage, unsigned char* d_expanded_key, int size) {
	int B = ceil((double)size / (512 * Nb));
	int T = 512;
	int threadN = B * T;
	decrypt <<<B, T >>> (d_bitmapImage, d_expanded_key, size, threadN);
}

__global__ void rsa_decrypt(int* cipher, unsigned char* expanded_key, int* rsa_private_key, int size) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < size) {
		int d = rsa_private_key[0];
		int p = rsa_private_key[1];
		int q = rsa_private_key[2];
		int n = p * q;
		int c = cipher[idx];
		int m = 1;
		for (int i = 0; i < d; ++i) {
			m = m * c;
			m = m % n;
		}
		expanded_key[idx] = (unsigned char)m;
	}
}

__declspec(dllexport) void launchRSADecryptKernel(int *cipher, unsigned char* d_expanded_key, int*rsa_private_key, int size) {
	int B = ceil((double)size / 128);
	rsa_decrypt <<<B, 128 >>> (cipher, d_expanded_key, rsa_private_key, size);
}
/*
int main()
{
	BITMAPINFOHEADER bitmapInfoHeader;
	BITMAPFILEHEADER bitmapFileHeader;
	unsigned char* bitmapData;
	unsigned char* d_bitmapImage;

	//////////////////////////////////////////////////////////////////////////////////////////// Expand key
	unsigned char key[16] = {
	0x2b, 0x28, 0xab, 0x09,
	0x7e, 0xae, 0xf7, 0xcf,
	0x15, 0xd2, 0x15, 0x4f,
	0x16, 0xa6, 0x88, 0x3c };
	// unsigned char key[] = "lqesutrlhajqzxck";
	unsigned char expanded_key[(Nr + 1) * Nb];
	key_expansion(key, expanded_key);
	unsigned char* d_expanded_key;
	hipMalloc((void**)&d_expanded_key, (Nr + 1) * Nb);
	hipMemcpy(d_expanded_key, expanded_key, (Nr + 1) * Nb, hipMemcpyHostToDevice);

	//////////////////////////////////////////////////////////////////////////////////////////// Encryption

	// Load image to CUDA memory
	bitmapData = LoadBitmapFile("lena.bmp", &bitmapInfoHeader, &bitmapFileHeader);
	hipMalloc((void**)&d_bitmapImage, bitmapInfoHeader.biSizeImage);
	hipMemcpy(d_bitmapImage, bitmapData, bitmapInfoHeader.biSizeImage, hipMemcpyHostToDevice);
	// Encryption kernel call
	int B = ceil(bitmapInfoHeader.biSizeImage / (512 * Nb));
	int T = 512;
	int threadN = B * T;
	encrypt << <B, T >> > (d_bitmapImage, d_expanded_key, bitmapInfoHeader.biSizeImage, threadN);
	// Save Encrypted image from CUDA memory to file
	hipMemcpy(bitmapData, d_bitmapImage, bitmapInfoHeader.biSizeImage, hipMemcpyDeviceToHost);
	SaveBitmapFile("Encrypted.bmp", bitmapData, &bitmapFileHeader, &bitmapInfoHeader);


	//////////////////////////////////////////////////////////////////////////////////////////// Decryption
	// load encrypted image from file tp CUDA memory
	bitmapData = LoadBitmapFile("Encrypted.bmp", &bitmapInfoHeader, &bitmapFileHeader);
	hipMemcpy(d_bitmapImage, bitmapData, bitmapInfoHeader.biSizeImage, hipMemcpyHostToDevice);
	// Decryption kernel call
	decrypt << <B, T >> > (d_bitmapImage, d_expanded_key, bitmapInfoHeader.biSizeImage, threadN);
	// Save Decrypted image from CUDA memory to file
	hipMemcpy(bitmapData, d_bitmapImage, bitmapInfoHeader.biSizeImage, hipMemcpyDeviceToHost);
	SaveBitmapFile("Decrypted.bmp", bitmapData, &bitmapFileHeader, &bitmapInfoHeader);

	hipFree(d_bitmapImage);
	hipFree(d_expanded_key);

	return 0;
} */